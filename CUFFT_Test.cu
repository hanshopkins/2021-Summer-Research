// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

// Complex data type
typedef float2 Complex;
static __host__ inline float ComplexAbs(Complex);

//This is the number of data points and stuff
#define N 50

constexpr double Pi = 3.14159265358979323846;

//this is the function we're transforming
float f(float t)
{
    return sin(2*Pi*t);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

    //memory for the function we're transforming
    Complex* h_fvalues = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * N));
    for (unsigned int i = 0; i < N; i++) //initializing
    {
        h_fvalues[i].x = f(i * 1.0 / N);
        h_fvalues[i].y = 0;
    }
    
    //device memory for the signal
    Complex* d_fvalues;
    hipMalloc(reinterpret_cast<void**>(&d_fvalues), sizeof(Complex)*N);
    //copy host memory to device
    hipMemcpy(d_fvalues, h_fvalues, sizeof(Complex)*N, hipMemcpyHostToDevice);

    //setting up the plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, sizeof(Complex)*N, HIPFFT_C2C,1);

    //execute plan. This transforms the signal in place.
    hipfftExecC2C(plan, d_fvalues, d_fvalues, HIPFFT_FORWARD);

    //copying the results back onto the host
    Complex* h_Fvalues = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * N));
    hipMemcpy(h_Fvalues, d_fvalues, sizeof(Complex) * N, hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < N; i++)
    {
         std::cout << "i: " << i << " Re(F): " << h_Fvalues[i].x/N << " Im(F): " << h_Fvalues[i].y/100 << " |F|: " << ComplexAbs(h_Fvalues[i])/100 << std::endl;
    }

    free(h_fvalues);
    hipFree(d_fvalues);
    free(h_Fvalues);
    return 0;
}

//Complex absolute value
static __host__ inline float ComplexAbs(Complex a)
{
    return sqrt(a.x * a.x + a.y * a.y);
}
